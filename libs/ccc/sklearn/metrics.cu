#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>
#include <assert.h>

// Todo: Add CudaCheckError

/**
 * @brief Unravel a flat index to the corresponding 2D indicis
 * @param[in] flat_idx The flat index to unravel
 * @param[in] num_cols Number of columns in the 2D array
 * @param[out] row Pointer to the row index
 * @param[out] col Pointer to the column index
 */
__device__ __host__ inline void unravel_index(int flat_idx, int num_cols, int *row, int *col)
{
    // change int to uint32_t
    *row = flat_idx / num_cols; // Compute row index
    *col = flat_idx % num_cols; // Compute column index
}

/**
 * @brief Given the number of objects and an index, this function calculates
 *        the coordinates in a symmetric matrix from a flat index.
 *        For example, if there are n_obj objects (such as genes), a condensed 
 *        1D array can be created with pairwise comparisons between these 
 *        objects, which corresponds to a symmetric 2D matrix. This function 
 *        calculates the 2D coordinates (x, y) in the symmetric matrix that 
 *        corresponds to the given flat index.
 *
 * @param[in] n_obj The total number of objects (i.e., the size of one dimension 
 *                  of the square symmetric matrix).
 * @param[in] idx The flat index from the condensed pairwise array.
 * @param[out] x Pointer to the calculated row coordinate in the symmetric matrix.
 * @param[out] y Pointer to the calculated column coordinate in the symmetric matrix.
 */
__device__ __host__ inline void get_coords_from_index(int n_obj, int idx, int *x, int *y)
{
    // Calculate 'b' based on the input n_obj
    int b = 1 - 2 * n_obj;
    // Calculate 'x' using the quadratic formula part
    float discriminant = b * b - 8 * idx;
    float x_float = floor((-b - sqrt(discriminant)) / 2);
    // Assign the integer part of 'x'
    *x = static_cast<int>(x_float);
    // Calculate 'y' based on 'x' and the index
    *y = static_cast<int>(idx + (*x) * (b + (*x) + 2) / 2 + 1);
}

/**
 * @brief Compute the contingency matrix for two partitions using shared memory
 * @param[in] part0 Pointer to the first partition array
 * @param[in] part1 Pointer to the second partition array
 * @param[in] n Number of elements in each partition array
 * @param[out] shared_cont_mat Pointer to shared memory for storing the contingency matrix
 * @param[in] k Maximum number of clusters (size of contingency matrix is k x k)
 */
__device__ void get_contingency_matrix(int *part0, int *part1, int n, int *shared_cont_mat, int k)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int num_threads = blockDim.x;
    int num_blocks = gridDim.x;
    int size = k * k;

    // Initialize shared memory
    for (int i = tid; i < size; i += num_threads)
    {
        shared_cont_mat[i] = 0;
    }
    __syncthreads();

    // Process elements
    for (int i = tid; i < n; i += num_threads)
    {
        int row = part0[i];
        int col = part1[i];

        if (row < k && col < k)
        {
            atomicAdd(&shared_cont_mat[row * k + col], 1);
        }
    }
    __syncthreads();
}


/**
 * @brief CUDA device function to compute the pair confusion matrix
 * @param[in] contingency Pointer to the contingency matrix
 * @param[in] sum_rows Pointer to the sum of rows in the contingency matrix
 * @param[in] sum_cols Pointer to the sum of columns in the contingency matrix
 * @param[in] n_objs Number of objects in each partition
 * @param[in] k Number of clusters (assuming k is the max of clusters in part0 and part1)
 * @param[out] C Pointer to the output pair confusion matrix (2x2)
 */
__device__ void get_pair_confusion_matrix(
    const int* __restrict__ contingency,
    int * sum_rows,
    int * sum_cols,
    const int n_objs,
    const int k,
    int* C
) {
    // Initialize sum_rows and sum_cols
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
        sum_rows[i] = 0;
        sum_cols[i] = 0;
    }
    __syncthreads();

    // Compute sum_rows and sum_cols
    for (int i = threadIdx.x; i < k * k; i += blockDim.x) {
        int row = i / k;
        int col = i % k;
        int val = contingency[i];
        atomicAdd(&sum_cols[col], val);
        atomicAdd(&sum_rows[row], val);
    }
    __syncthreads();
    
    // Compute sum_squares
    int sum_squares;
    if (threadIdx.x == 0) {
        sum_squares = 0;
        for (int i = 0; i < k * k; ++i) {
            sum_squares += (contingency[i]) * contingency[i];
        }
    }
    __syncthreads();
    // printf("sum_squares: %d\n", sum_squares);

    // Compute C[1,1], C[0,1], C[1,0], and C[0,0]
    if (threadIdx.x == 0) {
        C[3] = sum_squares - n_objs;  // C[1,1]

        int temp = 0;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < k; ++j) {
                temp += (contingency[i * k + j]) * sum_cols[j];
            }
        }
        C[1] = temp - sum_squares;  // C[0,1]

        temp = 0;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < k; ++j) {
                temp += (contingency[j * k + i]) * sum_rows[j];
            }
        }
        C[2] = temp - sum_squares;  // C[1,0]

        C[0] = n_objs * n_objs - C[1] - C[2] - sum_squares;  // C[0,0]

        // print C
        printf("C[0,0]: %d, C[0,1]: %d, C[1,0]: %d, C[1,1]: %d\n", C[0], C[1], C[2], C[3]);

        // compute ARI
        int tn = static_cast<float>(C[0]);
        int fp = static_cast<float>(C[1]);
        int fn = static_cast<float>(C[2]);
        int tp = static_cast<float>(C[3]);
        printf("tn: %d, fp: %d, fn: %d, tp: %d\n", tn, fp, fn, tp);
        float ari = 0.0;
        if (fn == 0 && fp ==0) {
            ari = 1.0;
        } else {
            ari = 2.0 * (tp * tn - fn * fp) / ((tp + fn) * (fn + tn) + (tp + fp) * (fp + tn));
        }
        printf("ari: %f\n", ari);
    }
}

/**
 * @brief Main ARI kernel. Now only compare a pair of ARIs
 * @param n_parts Number of partitions of each feature
 * @param n_objs Number of objects in each partitions
 * @param n_part_mat_elems Number of elements in the square partition matrix
 * @param n_elems_per_feat Number of elements for each feature, i.e., part[i].x * part[i].y
 * @param parts 3D Array of partitions with shape of (n_features, n_parts, n_objs)
 * @param n_aris Number of ARIs to compute
 * @param k The max value of cluster number + 1
 * @param out Output array of ARIs
 * @param part_pairs Output array of part pairs to be compared by ARI
 */
__global__ void ari(int *parts,
                    const int n_aris,
                    const int n_features,
                    const int n_parts,
                    const int n_objs,
                    const int n_elems_per_feat,
                    const int n_part_mat_elems,
                    const int k,
                    float *out,
                    int *part_pairs = nullptr)
{
    /*
     * Step 1: Each thead, unravel flat indices and load the corresponding data into shared memory
     */
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    // each block is responsible for one ARI computation
    int ari_block_idx = blockIdx.x;

    // obtain the corresponding parts and unique counts
    int feature_comp_flat_idx = ari_block_idx / n_part_mat_elems; // flat comparison pair index for two features
    int part_pair_flat_idx = ari_block_idx % n_part_mat_elems;    // flat comparison pair index for two partitions of one feature pair
    int i, j;

    // unravel the feature indices
    get_coords_from_index(n_features, feature_comp_flat_idx, &i, &j);
    assert(i < n_features && j < n_features);
    assert(i >= 0 && j >= 0);

    // unravel the partition indices
    int m, n;
    unravel_index(part_pair_flat_idx, n_parts, &m, &n);

    // Make pointers to select the parts and unique counts for the feature pair
    // Todo: Use int4*?
    int *t_data_part0 = parts + i * n_elems_per_feat + m * n_objs; // t_ for thread
    int *t_data_part1 = parts + j * n_elems_per_feat + n * n_objs;

    // Load gmem data into smem by using different threads
    extern __shared__ int shared_mem[];
    int *s_part0 = shared_mem;
    int *s_part1 = shared_mem + n_objs;

    // Loop over the data using the block-stride pattern
    for (int i = threadIdx.x; i < n_objs; i += blockDim.x)
    {
        s_part0[i] = t_data_part0[i];
        s_part1[i] = t_data_part1[i];
    }
    __syncthreads();

    // Copy data to global memory if part_pairs is specified
    if (part_pairs != nullptr)
    {
        int *out_part0 = part_pairs + ari_block_idx * (2 * n_objs);
        int *out_part1 = out_part0 + n_objs;

        for (int i = threadIdx.x; i < n_objs; i += blockDim.x)
        {
            out_part0[i] = s_part0[i];
            out_part1[i] = s_part1[i];
        }
    }

    /*
     * Step 2: Compute contingency matrix within the block
     */
    // shared mem address for the contingency matrix
    int *s_contingency = shared_mem + 2 * n_objs;
    get_contingency_matrix(t_data_part0, t_data_part1, n_objs, s_contingency, k);

    /*
     * Step 3: Construct pair confusion matrix
     */
    // shared mem address for the pair confusion matrix
    int *s_sum_rows = s_contingency + k * k;
    int *s_sum_cols = s_sum_rows + k;
    int *s_pair_confusion_matrix = s_sum_cols + k;
    get_pair_confusion_matrix(s_contingency, s_sum_rows, s_sum_cols, n_objs, k, s_pair_confusion_matrix);
    /*
     * Step 4: Compute ARI and write to global memory
     */
    if (threadIdx.x == 0) {
        int tn = static_cast<float>(s_pair_confusion_matrix[0]);
        int fp = static_cast<float>(s_pair_confusion_matrix[1]);
        int fn = static_cast<float>(s_pair_confusion_matrix[2]);
        int tp = static_cast<float>(s_pair_confusion_matrix[3]);
        printf("tn: %d, fp: %d, fn: %d, tp: %d\n", tn, fp, fn, tp);
        float ari = 0.0;
        if (fn == 0 && fp == 0) {
            ari = 1.0;
        } else {
            ari = 2.0 * (tp * tn - fn * fp) / ((tp + fn) * (fn + tn) + (tp + fp) * (fp + tn));
        }
        printf("ari: %f\n", ari);
        out[ari_block_idx] = ari;
    }
    __syncthreads();

}

// Helper function to generate pairwise combinations (implement this according to your needs)
std::vector<std::pair<std::vector<int>, std::vector<int>>> generate_pairwise_combinations(const std::vector<std::vector<std::vector<int>>> &arr)
{
    std::vector<std::pair<std::vector<int>, std::vector<int>>> pairs;
    size_t num_slices = arr.size(); // Number of 2D arrays in the 3D vector
    for (size_t i = 0; i < num_slices; ++i)
    {
        for (size_t j = i + 1; j < num_slices; ++j)
        { // Only consider pairs in different slices
            for (const auto &row_i : arr[i])
            { // Each row in slice i
                for (const auto &row_j : arr[j])
                { // Pairs with each row in slice j
                    pairs.emplace_back(row_i, row_j);
                }
            }
        }
    }
    return pairs;
}

void test_ari_parts_selection()
{
    // Define test input
    std::vector<std::vector<std::vector<int>>> parts = {
        {{0, 1, 2, 3},
         {0, 2, 3, 4},
         {0, 3, 4, 5}},
        {{1, 1, 2, 3},
         {1, 2, 3, 4},
         {1, 3, 4, 5}},
        {{2, 1, 2, 3},
         {2, 2, 3, 4},
         {2, 3, 4, 5}}};

    const int k = 6; // specified by the call to ccc , part number from [0...9]
    
    // std::vector<std::vector<std::vector<int>>> parts = {
    //     {{4, 1, 3, 5, 2, 0, 6, 3, 1, 4},
    //     {0, 2, 6, 4, 5, 3, 1, 0, 6, 2},
    //     {1, 5, 3, 2, 4, 0, 6, 1, 5, 3}},
        
    //     // {{3, 6, 0, 2, 1, 5, 4, 3, 6, 0},
    //     // {5, 1, 4, 0, 3, 6, 2, 1, 5, 4},
    //     // {2, 3, 6, 1, 0, 5, 4, 3, 6, 2}},
        
    //     {{1, 4, 5, 3, 6, 0, 2, 5, 4, 1},
    //     {0, 6, 2, 5, 1, 3, 4, 6, 0, 2},
    //     {4, 1, 3, 6, 5, 0, 2, 4, 1, 3}}
    // };

    // const int k = 7; // specified by the call to ccc , max(parts) + 1
    

    // std::vector<int> part_maxes = {3, 4, 5, 3, 4, 5, 3, 4, 5};
    // auto sz_part_maxes = sizeof(part_maxes) / sizeof(part_maxes[0]);

    // Get dimensions
    int n_features = parts.size();
    int n_parts = parts[0].size();
    int n_objs = parts[0][0].size();
    int n_feature_comp = n_features * (n_features - 1) / 2;
    int n_aris = n_feature_comp * n_parts * n_parts;
    std::cout << "n_features: " << n_features << ", n_parts: " << n_parts << ", n_objs: " << n_objs << std::endl
              << "n_feature_comps: " << n_feature_comp << ", n_aris: " << n_aris << std::endl;

    // Allocate host memory for C-style array
    int *h_parts = new int[n_features * n_parts * n_objs];

    // Copy data from vector to C-style array
    for (int i = 0; i < n_features; ++i)
    {
        for (int j = 0; j < n_parts; ++j)
        {
            for (int k = 0; k < n_objs; ++k)
            {
                h_parts[i * (n_parts * n_objs) + j * n_objs + k] = parts[i][j][k];
            }
        }
    }

    // Set up CUDA kernel configuration
    int block_size = 2;
    // Each block is responsible for one ARI computation
    int grid_size = n_aris;
    // Compute shared memory size
    size_t s_mem_size = n_objs * 2 * sizeof(int); // For the partition pair to be compared
    s_mem_size += 2 * k * sizeof(int); // For the internal sum arrays
    s_mem_size += 4 * sizeof(int); // For the 2 x 2 confusion matrix

    // Allocate device memory
    int *d_parts, *d_parts_pairs;
    float *d_out;
    hipMalloc(&d_parts, n_features * n_parts * n_objs * sizeof(int));
    hipMalloc(&d_out, n_aris * sizeof(float));
    hipMalloc(&d_parts_pairs, n_aris * 2 * n_objs * sizeof(int));

    // Copy data to device
    hipMemcpy(d_parts, h_parts, n_features * n_parts * n_objs * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    ari<<<grid_size, block_size, s_mem_size>>>(
        d_parts,
        n_aris,
        n_features,
        n_parts,
        n_objs,
        n_parts * n_objs,
        n_parts * n_parts,
        k,
        d_out,
        d_parts_pairs);

    // Synchronize device
    hipDeviceSynchronize();

    // Copy results back to host
    int *h_parts_pairs = new int[n_aris * 2 * n_objs];
    hipMemcpy(h_parts_pairs, d_parts_pairs, n_aris * 2 * n_objs * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Parts pairs: " << std::endl;
    for (int i = 0; i < n_aris; ++i)
    {
        std::cout << "Pair:" << i << std::endl;
        for (int j = 0; j < 2; ++j)
        {
            for (int k = 0; k < n_objs; ++k)
            {
                std::cout << *(h_parts_pairs + i * 2 * n_objs + j * n_objs + k) << ", ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;

    // Assert equality on the parts pairs
    bool all_equal = true;
    auto pairs = generate_pairwise_combinations(parts);
    int n_pairs = pairs.size();
    for (int i = 0; i < n_pairs; ++i)
    {
        for (int j = 0; j < 2; ++j)
        {
            const std::vector<int> &current_vector = (j == 0) ? pairs[i].first : pairs[i].second;
            for (int k = 0; k < n_objs; ++k)
            {
                int flattened_index = i * 2 * n_objs + j * n_objs + k;
                if (h_parts_pairs[flattened_index] != current_vector[k])
                {
                    all_equal = false;
                    std::cout << "Mismatch at i=" << i << ", j=" << j << ", k=" << k << std::endl;
                    std::cout << "Expected: " << current_vector[k] << ", Got: " << h_parts_pairs[flattened_index] << std::endl;
                }
            }
        }
    }

    if (all_equal)
    {
        std::cout << "Test passed: All elements match." << std::endl;
    }
    else
    {
        std::cout << "Test failed: Mismatches found." << std::endl;
    }

    // Print ARI results
    float *h_out = new float[n_aris];
    hipMemcpy(h_out, d_out, n_aris * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "ARI results: " << std::endl;
    for (int i = 0; i < n_aris; ++i)
    {
        printf("%f, ", h_out[i]);
    }
    std::cout << std::endl;

    // Clean up
    hipFree(d_parts);
    hipFree(d_out);
    hipFree(d_parts_pairs);
    delete[] h_parts_pairs;
}

int main()
{
    test_ari_parts_selection();
    return 0;
}