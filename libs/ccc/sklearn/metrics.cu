#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>
#include <assert.h>

// #define N_OBJS 16
// #define N_PARTS 1
// #define N_FEATURES 2

/**
 * @brief Unravel a flat index to the corresponding 2D indicis
 * @param[in] flat_idx The flat index to unravel
 * @param[in] num_cols Number of columns in the 2D array
 * @param[out] row Pointer to the row index
 * @param[out] col Pointer to the column index
 */
__device__ __host__ inline void unravel_index(int flat_idx, int num_cols, int *row, int *col)
{
    // change int to uint32_t
    *row = flat_idx / num_cols; // Compute row index
    *col = flat_idx % num_cols; // Compute column index
}

__device__ __host__ inline void get_coords_from_index(int n_obj, int idx, int *x, int *y)
{
    // Calculate 'b' based on the input n_obj
    int b = 1 - 2 * n_obj;
    // Calculate 'x' using the quadratic formula part
    float discriminant = b * b - 8 * idx;
    float x_float = floor((-b - sqrt(discriminant)) / 2);
    // Assign the integer part of 'x'
    *x = static_cast<int>(x_float);
    // Calculate 'y' based on 'x' and the index
    *y = static_cast<int>(idx + (*x) * (b + (*x) + 2) / 2 + 1);
}

/**
 * @brief Compute the contingency matrix for two partitions using shared memory
 * @param[in] part0 Pointer to the first partition array
 * @param[in] part1 Pointer to the second partition array
 * @param[in] n Number of elements in each partition array
 * @param[out] shared_cont_mat Pointer to shared memory for storing the contingency matrix
 * @param[in] k Maximum number of clusters (size of contingency matrix is k x k)
 */
__device__ void get_contingency_matrix(int *part0, int *part1, int n, int *shared_cont_mat, int k)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int num_threads = blockDim.x;
    int num_blocks = gridDim.x;
    int size = k * k;

    // Initialize shared memory
    for (int i = tid; i < size; i += num_threads)
    {
        shared_cont_mat[i] = 0;
    }
    __syncthreads();

    // Process elements
    for (int i = tid; i < n; i += num_threads)
    {
        int row = part0[i];
        int col = part1[i];

        if (row < k && col < k)
        {
            atomicAdd(&shared_cont_mat[row * k + col], 1);
        }
    }
    __syncthreads();
}


/**
 * @brief CUDA device function to compute the pair confusion matrix
 * @param[in] contingency Pointer to the contingency matrix
 * @param[in] sum_rows Pointer to the sum of rows in the contingency matrix
 * @param[in] sum_cols Pointer to the sum of columns in the contingency matrix
 * @param[in] n_objs Number of objects in each partition
 * @param[in] k Number of clusters (assuming k is the max of clusters in part0 and part1)
 * @param[out] C Pointer to the output pair confusion matrix (2x2)
 */
__device__ void get_pair_confusion_matrix(
    const int* __restrict__ contingency,
    int * sum_rows,
    int * sum_cols,
    const int n_objs,
    const int k,
    int* C
) {
    // Initialize sum_rows and sum_cols
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
        sum_rows[i] = 0;
        sum_cols[i] = 0;
    }
    __syncthreads();

    // Compute sum_rows and sum_cols
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
        for (int m = 0; m < k; ++m) {
            for (int n = 0; n < k; ++n) {
                const int val = contingency[m * k + n];
                atomicAdd(&sum_rows[m], val);
                atomicAdd(&sum_cols[n], val);
            }
        }
    }
    __syncthreads();

    // Compute sum_squares
    int sum_squares;
    if (threadIdx.x == 0) {
        sum_squares = 0;
        for (int i = 0; i < k * k; ++i) {
            sum_squares += (contingency[i]) * contingency[i];
        }
    }
    __syncthreads();

    // Compute C[1,1], C[0,1], C[1,0], and C[0,0]
    if (threadIdx.x == 0) {
        C[3] = sum_squares - n_samples;  // C[1,1]

        long long temp = 0;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < k; ++j) {
                temp += static_cast<long long>(contingency[i * k + j]) * sum0[j];
            }
        }
        C[1] = temp - sum_squares;  // C[0,1]

        temp = 0;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < k; ++j) {
                temp += static_cast<long long>(contingency[j * k + i]) * sum1[j];
            }
        }
        C[2] = temp - sum_squares;  // C[1,0]

        C[0] = static_cast<long long>(n_samples) * n_samples - C[1] - C[2] - sum_squares;  // C[0,0]
    }
}

/**
 * @brief Main ARI kernel. Now only compare a pair of ARIs
 * @param n_parts Number of partitions of each feature
 * @param n_objs Number of objects in each partitions
 * @param n_part_mat_elems Number of elements in the square partition matrix
 * @param n_elems_per_feat Number of elements for each feature, i.e., part[i].x * part[i].y
 * @param parts 3D Array of partitions with shape of (n_features, n_parts, n_objs)
 * @param n_aris Number of ARIs to compute
 * @param out Output array of ARIs
 * @param part_pairs Output array of part pairs to be compared by ARI
 */
__global__ void ari(int *parts,
                    const int n_aris,
                    const int n_features,
                    const int n_parts,
                    const int n_objs,
                    const int n_elems_per_feat,
                    const int n_part_mat_elems,
                    const int k,
                    float *out,
                    int *part_pairs = nullptr)
{
    /*
     * Step 1: Each thead, unravel flat indices and load the corresponding data into shared memory
     */
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    // each block is responsible for one ARI computation
    int ari_block_idx = blockIdx.x;

    // print parts for debugging
    if (global_tid == 0)
    {
        for (int i = 0; i < n_features; ++i)
        {
            for (int j = 0; j < n_parts; ++j)
            {
                for (int k = 0; k < n_objs; ++k)
                {
                    printf("parts[%d][%d][%d]: %d\n", i, j, k, parts[i * n_parts * n_objs + j * n_objs + k]);
                }
            }
            printf("\n");
        }
    }

    // obtain the corresponding parts and unique counts
    printf("n_part_mat_elems: %d\n", n_part_mat_elems);
    int feature_comp_flat_idx = ari_block_idx / n_part_mat_elems; // flat comparison pair index for two features
    int part_pair_flat_idx = ari_block_idx % n_part_mat_elems;    // flat comparison pair index for two partitions of one feature pair
    int i, j;

    if (global_tid == 0)
    {
        printf("ari_block_idx: %d, feature_comp_flat_idx: %d, part_pair_flat_idx: %d\n", ari_block_idx, feature_comp_flat_idx, part_pair_flat_idx);
    }

    // unravel the feature indices
    get_coords_from_index(n_features, feature_comp_flat_idx, &i, &j);
    assert(i < n_features && j < n_features);
    assert(i >= 0 && j >= 0);
    if (global_tid == 0)
    {
        printf("global_tid: %d, i: %d, j: %d\n", global_tid, i, j);
    }
    // unravel the partition indices
    int m, n;
    unravel_index(part_pair_flat_idx, n_parts, &m, &n);
    if (global_tid == 0)
    {
        printf("global_tid: %d, m: %d, n: %d\n", global_tid, m, n);
    }

    // Make pointers to select the parts and unique counts for the feature pair
    // Todo: Use int4*?
    int *t_data_part0 = parts + i * n_elems_per_feat + m * n_objs; // t_ for thread
    int *t_data_part1 = parts + j * n_elems_per_feat + n * n_objs;

    // Load gmem data into smem by using different threads
    extern __shared__ int shared_mem[];
    int *s_part0 = shared_mem;
    int *s_part1 = shared_mem + n_objs;

    // Loop over the data using the block-stride pattern
    for (int i = threadIdx.x; i < n_objs; i += blockDim.x)
    {
        s_part0[i] = t_data_part0[i];
        s_part1[i] = t_data_part1[i];
    }
    __syncthreads();

    // Copy data to global memory if part_pairs is specified
    if (part_pairs != nullptr)
    {
        int *out_part0 = part_pairs + ari_block_idx * (2 * n_objs);
        int *out_part1 = out_part0 + n_objs;

        for (int i = threadIdx.x; i < n_objs; i += blockDim.x)
        {
            out_part0[i] = s_part0[i];
            out_part1[i] = s_part1[i];
        }
    }

    /*
     * Step 2: Compute contingency matrix within the block
     */
    // shared mem address for the contingency matrix
    int *s_contingency = shared_mem + 2 * n_objs;
    // initialize the contingency matrix to zero
    // const int n_contingency_items = k * k;
    // for (int i = threadIdx.x; i < n_contingency_items; i += blockDim.x) {
    //     s_contingency[i] = 0;
    // }
    get_contingency_matrix(t_data_part0, t_data_part1, n_objs, s_contingency, k);
    if (global_tid == 0)
    {
        for (int i = 0; i < k; ++i)
        {
            for (int j = 0; j < k; ++j)
            {
                printf("s_contingency[%d][%d]: %d\n", i, j, s_contingency[i * k + j]);
            }
        }
    }

    /*
     * Step 3: Construct pair confusion matrix
     */
    // shared mem address for the pair confusion matrix
    int *s_sum_rows = s_contingency + k * k;
    int *s_sum_cols = s_sum_rows + k;
    int *s_pair_confusion_matrix = s_sum_cols + k;
    
    /*
     * Step 4: Compute ARI and write to global memory
     */
}

// Helper function to generate pairwise combinations (implement this according to your needs)
std::vector<std::pair<std::vector<int>, std::vector<int>>> generate_pairwise_combinations(const std::vector<std::vector<std::vector<int>>> &arr)
{
    std::vector<std::pair<std::vector<int>, std::vector<int>>> pairs;
    size_t num_slices = arr.size(); // Number of 2D arrays in the 3D vector
    for (size_t i = 0; i < num_slices; ++i)
    {
        for (size_t j = i + 1; j < num_slices; ++j)
        { // Only consider pairs in different slices
            for (const auto &row_i : arr[i])
            { // Each row in slice i
                for (const auto &row_j : arr[j])
                { // Pairs with each row in slice j
                    pairs.emplace_back(row_i, row_j);
                }
            }
        }
    }
    return pairs;
}

void test_ari_parts_selection()
{
    // Define test input
    std::vector<std::vector<std::vector<int>>> parts = {
        {{0, 1, 2, 3},
         {0, 2, 3, 4},
         {0, 3, 4, 5}},
        {{1, 1, 2, 3},
         {1, 2, 3, 4},
         {1, 3, 4, 5}},
        {{2, 1, 2, 3},
         {2, 2, 3, 4},
         {2, 3, 4, 5}}};

    const int k = 6; // specified by the call to ccc , part number from [0...9]
    // std::vector<int> part_maxes = {3, 4, 5, 3, 4, 5, 3, 4, 5};
    // auto sz_part_maxes = sizeof(part_maxes) / sizeof(part_maxes[0]);

    // Get dimensions
    int n_features = parts.size();
    int n_parts = parts[0].size();
    int n_objs = parts[0][0].size();
    int n_feature_comp = n_features * (n_features - 1) / 2;
    int n_aris = n_feature_comp * n_parts * n_parts;
    std::cout << "n_features: " << n_features << ", n_parts: " << n_parts << ", n_objs: " << n_objs << std::endl
              << "n_feature_comps: " << n_feature_comp << ", n_aris: " << n_aris << std::endl;

    // Allocate host memory for C-style array
    int *h_parts = new int[n_features * n_parts * n_objs];

    // Copy data from vector to C-style array
    for (int i = 0; i < n_features; ++i)
    {
        for (int j = 0; j < n_parts; ++j)
        {
            for (int k = 0; k < n_objs; ++k)
            {
                h_parts[i * (n_parts * n_objs) + j * n_objs + k] = parts[i][j][k];
            }
        }
    }

    // Set up CUDA kernel configuration
    int block_size = 2;
    // Each block is responsible for one ARI computation
    int grid_size = n_aris;
    // Compute shared memory size
    size_t s_mem_size = n_objs * 2 * sizeof(int); // For the partition pair to be compared
    s_mem_size += 2 * k * sizeof(int); // For the internal sum arrays
    s_mem_size += 4 * sizeof(int); // For the 2 x 2 confusion matrix

    // Allocate device memory
    int *d_parts, *d_parts_pairs;
    float *d_out;
    hipMalloc(&d_parts, n_features * n_parts * n_objs * sizeof(int));
    hipMalloc(&d_out, n_aris * sizeof(float));
    hipMalloc(&d_parts_pairs, n_aris * 2 * n_objs * sizeof(int));

    // Copy data to device
    hipMemcpy(d_parts, h_parts, n_features * n_parts * n_objs * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    ari<<<grid_size, block_size, s_mem_size>>>(
        d_parts,
        n_aris,
        n_features,
        n_parts,
        n_objs,
        n_parts * n_objs,
        n_parts * n_parts,
        k,
        d_out,
        d_parts_pairs);

    // Synchronize device
    hipDeviceSynchronize();

    // Copy results back to host
    int *h_parts_pairs = new int[n_aris * 2 * n_objs];
    hipMemcpy(h_parts_pairs, d_parts_pairs, n_aris * 2 * n_objs * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Parts pairs: " << std::endl;
    for (int i = 0; i < n_aris; ++i)
    {
        std::cout << "Pair:" << i << std::endl;
        for (int j = 0; j < 2; ++j)
        {
            for (int k = 0; k < n_objs; ++k)
            {
                std::cout << *(h_parts_pairs + i * 2 * n_objs + j * n_objs + k) << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl
                  << std::endl;
    }
    std::cout << std::endl;

    // Assert equality on the parts pairs
    bool all_equal = true;
    auto pairs = generate_pairwise_combinations(parts);
    int n_pairs = pairs.size();
    for (int i = 0; i < n_pairs; ++i)
    {
        for (int j = 0; j < 2; ++j)
        {
            const std::vector<int> &current_vector = (j == 0) ? pairs[i].first : pairs[i].second;
            for (int k = 0; k < n_objs; ++k)
            {
                int flattened_index = i * 2 * n_objs + j * n_objs + k;
                if (h_parts_pairs[flattened_index] != current_vector[k])
                {
                    all_equal = false;
                    std::cout << "Mismatch at i=" << i << ", j=" << j << ", k=" << k << std::endl;
                    std::cout << "Expected: " << current_vector[k] << ", Got: " << h_parts_pairs[flattened_index] << std::endl;
                }
            }
        }
    }

    if (all_equal)
    {
        std::cout << "Test passed: All elements match." << std::endl;
    }
    else
    {
        std::cout << "Test failed: Mismatches found." << std::endl;
    }

    // Clean up
    hipFree(d_parts);
    hipFree(d_out);
    hipFree(d_parts_pairs);
    delete[] h_parts_pairs;
}

int main()
{
    test_ari_parts_selection();
    return 0;
}